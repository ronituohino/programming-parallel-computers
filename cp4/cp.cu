#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(x) check(x, #x)
static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

__global__ void compute_means_and_normalize(int ny, int nx, const float *data, float *normal, float *nss)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y >= ny)
  {
    return;
  }

  float sum = 0.0;
  for (int x = 0; x < nx; ++x)
  {
    sum += data[y * nx + x];
  }
  float mean = sum / nx;

  float pow_sum = 0.0;
  for (int x = 0; x < nx; ++x)
  {
    float normalized = data[y * nx + x] - mean;
    normal[y * nx + x] = normalized;
    pow_sum += normalized * normalized;
  }
  nss[y] = sqrt(pow_sum);
}

__global__ void compute_correlations(int ny, int nx, const float *normal, const float *nss, float *result)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= ny || j >= ny || j < i)
    return;

  float sum = 0.0;
  for (int n = 0; n < nx; n++)
  {
    sum += normal[i * nx + n] * normal[j * nx + n];
  }

  result[j + i * ny] = sum / (nss[i] * nss[j]);
}

void correlate(int ny, int nx, const float *data, float *result)
{
  // Allocate device memory
  float *d_data, *d_normal, *d_nss, *d_result;

  size_t data_size = ny * nx * sizeof(float);
  size_t normal_size = ny * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);
  size_t nss_size = ny * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_normal, normal_size));
  CHECK(hipMalloc((void **)&d_nss, nss_size));
  CHECK(hipMalloc((void **)&d_result, result_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_normal, 0, normal_size));
  CHECK(hipMemset(d_nss, 0, nss_size));
  CHECK(hipMemset(d_result, 0, result_size));

  // Copy input data to device
  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  // Launch kernel to compute means and normalize data
  {
    dim3 grid_dim(divup(ny, 32));
    dim3 block_dim(32); // Make amount of threads a multiple of 32 (GPU warp size)
    compute_means_and_normalize<<<grid_dim, block_dim>>>(ny, nx, d_data, d_normal, d_nss);
    CHECK(hipGetLastError());
  }

  // Launch kernel to compute correlations
  {
    dim3 grid_dim(divup(ny, 32), divup(ny, 32), 1);
    dim3 block_dim(32, 32, 1); // Make amount of threads a multiple of 32 (GPU warp size)
    compute_correlations<<<grid_dim, block_dim>>>(ny, nx, d_normal, d_nss, d_result);
    CHECK(hipGetLastError());
  }

  // Copy result back to host
  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  // Free device memory
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_normal));
  CHECK(hipFree(d_nss));
  CHECK(hipFree(d_result));
}
