#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <iostream>

#define CHECK(x) check(x, #x)
static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__global__ void compute_means(int ny, int nx, const float *data, float *means)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int y = tid; y < ny; y += stride)
  {
    float sum = 0.0f;
    for (int x = 0; x < nx; ++x)
    {
      sum += data[y * nx + x];
    }
    means[y] = sum / nx;
  }
}

__global__ void normalize_and_compute_pow_sums(int ny, int nx, const float *data, const float *means, float *norm_data, float *pow_sums)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int y = tid; y < ny; y += stride)
  {
    float pow_sum = 0.0f;
    for (int x = 0; x < nx; ++x)
    {
      float norm = data[y * nx + x] - means[y];
      norm_data[y * nx + x] = norm;
      pow_sum += norm * norm;
    }
    pow_sums[y] = pow_sum;
  }
}

__global__ void compute_correlations(int ny, int nx, const float *norm_data, const float *pow_sums, float *result)
{
  int i = blockIdx.x;
  int j = threadIdx.x;

  if (i < ny && j < i)
  {
    float sum = 0.0f;
    for (int x = 0; x < nx; ++x)
    {
      sum += norm_data[i * nx + x] * norm_data[j * nx + x];
    }
    float inv_sqrt_i = rsqrtf(pow_sums[i]);
    float inv_sqrt_j = rsqrtf(pow_sums[j]);
    result[i * ny + j] = sum * inv_sqrt_i * inv_sqrt_j;
  }
}

void correlate(int ny, int nx, const float *data, float *result)
{
  // Allocate GPU memory
  float *d_data, *d_means, *d_norm_data, *d_pow_sums, *d_result;

  size_t data_size = ny * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_means, ny * sizeof(float)));
  CHECK(hipMalloc((void **)&d_norm_data, data_size));
  CHECK(hipMalloc((void **)&d_pow_sums, ny * sizeof(float)));
  CHECK(hipMalloc((void **)&d_result, result_size));

  // Copy data
  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  int block_size = 256;
  int grid_size = (ny + block_size - 1) / block_size;

  compute_means<<<grid_size, block_size>>>(ny, nx, d_data, d_means);
  CHECK(hipGetLastError());

  normalize_and_compute_pow_sums<<<grid_size, block_size>>>(ny, nx, d_data, d_means, d_norm_data, d_pow_sums);
  CHECK(hipGetLastError());

  dim3 grid_dim(ny, 1, 1);
  dim3 block_dim(ny, 1, 1);
  compute_correlations<<<grid_dim, block_dim>>>(ny, nx, d_norm_data, d_pow_sums, d_result);
  CHECK(hipGetLastError());

  // Get results
  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  // Free GPU memory
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_means));
  CHECK(hipFree(d_norm_data));
  CHECK(hipFree(d_pow_sums));
  CHECK(hipFree(d_result));
}