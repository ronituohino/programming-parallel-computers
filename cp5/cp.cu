#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(x) check(x, #x)
static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

// Returns the minimum units (cast to int to floor) to fill a, with b -size parts
static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

__global__ void compute_means_and_normalize(int ny, int nx, int nyp, int nxp, const float *data, float *norm, float *nss)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y >= ny)
  {
    return;
  }

  float sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    sum += data[y * nx + x];
  }
  float mean = sum / nx;

  float pow_sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    float normalized = data[y * nx + x] - mean;
    norm[y * nxp + x] = normalized;
    pow_sum += normalized * normalized;
  }
  nss[y] = 1.0 / sqrt(pow_sum);
}

__global__ void compute_correlations(int ny, int nx, int nyp, int nxp, int y_parts, int x_parts, const float *norm, float *par_res)
{
  int x0 = blockIdx.x;
  int y0 = blockIdx.y;
  int y1 = blockIdx.z;

  if (y1 < y0)
  {
    return;
  }

  float sums[16] = {
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0};

  for (int x = 0; x < nxp / x_parts; x++)
  {
    for (int n = 0; n < 16; n++)
    {
      int nx_idx = n / 4;
      int ns_idx = n % 4;

      int i = y0 * 4 + nx_idx;
      int j = y1 * 4 + ns_idx;

      if (i < ny && j < ny && (x + x0 * (nxp / x_parts)) < nx)
      {
        float a = norm[i * nxp + x + x0 * (nxp / x_parts)];
        float b = norm[j * nxp + x + x0 * (nxp / x_parts)];
        sums[n] += a * b;
      }
    }
  }

  // Write out sums to partial results
  int addr = (x0 * y_parts * y_parts * 16) + (y0 * y_parts + y1) * 16;
  for (int n = 0; n < 16; n++)
  {
    par_res[addr + n] = sums[n];
  }
}

__global__ void compute_sums(int ny, int nx, int nyp, int nxp, int y_parts, int x_parts, const float *par_res, const float *nss, float *result)
{
  int y0 = blockIdx.y;
  int y1 = blockIdx.z;

  if (y1 < y0)
  {
    return;
  }

  float sums[16] = {
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0};

  for (int x0 = 0; x0 < x_parts; x0++)
  {
    int addr = (x0 * y_parts * y_parts * 16) + (y0 * y_parts + y1) * 16;

    // Fetch all values from partial results and calulate total sum for these rows
    for (int n = 0; n < 16; n++)
    {
      sums[n] += par_res[addr + n];
    }
  }

  for (int n = 0; n < 16; n++)
  {
    int nx_idx = n / 4;
    int ns_idx = n % 4;

    int i = y0 * 4 + nx_idx;
    int j = y1 * 4 + ns_idx;

    if (i < ny && j < ny)
    {
      result[i * ny + j] = sums[n] * (nss[i] * nss[j]);
    }
  }
}

void correlate(int ny, int nx, const float *data, float *result)
{
  int y_parts = divup(ny, 4);
  int nyp = y_parts * 4;

  int x_parts = divup(nx, 4);
  int nxp = x_parts * 4;

  // Allocate device memory
  float *d_data, *d_norm, *d_nss, *d_par_res, *d_result;

  size_t data_size = ny * nx * sizeof(float);
  size_t norm_size = nyp * nxp * sizeof(float);
  size_t nss_size = ny * sizeof(float);
  size_t par_res_size = x_parts * y_parts * y_parts * 16 * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_norm, norm_size));
  CHECK(hipMalloc((void **)&d_nss, nss_size));
  CHECK(hipMalloc((void **)&d_par_res, par_res_size));
  CHECK(hipMalloc((void **)&d_result, result_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_norm, 0, norm_size));
  CHECK(hipMemset(d_nss, 0, nss_size));
  CHECK(hipMemset(d_par_res, 0, par_res_size));
  CHECK(hipMemset(d_result, 0, result_size));

  // Copy input data to device
  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  // Launch kernel to compute means and normalize data
  {
    dim3 grid(divup(nyp, 8));
    dim3 block(8);
    compute_means_and_normalize<<<grid, block>>>(ny, nx, nyp, nxp, d_data, d_norm, d_nss);
    CHECK(hipGetLastError());
  }

  // Launch kernel to compute correlations
  {
    dim3 grid(x_parts, y_parts, y_parts);
    dim3 block(1);
    compute_correlations<<<grid, block>>>(ny, nx, nyp, nxp, y_parts, x_parts, d_norm, d_par_res);
    CHECK(hipGetLastError());
  }

  // Launch kernel to compute final sums
  {
    dim3 grid(1, y_parts, y_parts);
    dim3 block(1);
    compute_sums<<<grid, block>>>(ny, nx, nyp, nxp, y_parts, x_parts, d_par_res, d_nss, d_result);
    CHECK(hipGetLastError());
  }

  // Copy result back to host
  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  // Free device memory
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_norm));
  CHECK(hipFree(d_nss));
  CHECK(hipFree(d_par_res));
  CHECK(hipFree(d_result));
}
