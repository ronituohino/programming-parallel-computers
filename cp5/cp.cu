#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(x) check(x, #x)
static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

__global__ void compute_means_and_normalize(int ny, int nx, const float *data, float *normal, float *nss)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y >= ny)
  {
    return;
  }

  float sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    sum += data[y * nx + x];
  }
  float mean = sum / nx;

  float pow_sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    float normalized = data[y * nx + x] - mean;
    normal[y * nx + x] = normalized;
    pow_sum += normalized * normalized;
  }
  nss[y] = 1.0 / sqrt(pow_sum);
}

__global__ void compute_correlations(int ny, int nx, const float *normal, const float *nss, float *result)
{
  int i_dx = blockIdx.x * blockDim.x + threadIdx.x;
  int j_dx = blockIdx.y * blockDim.y + threadIdx.y;

  if (j_dx < i_dx)
  {
    return;
  }

  int is = i_dx * 8;
  int js = j_dx * 8;

  float sums[64] = {0.0};
  for (int x = 0; x < nx; x++)
  {
    for (int ii = 0; ii < 8; ii++)
    {
      for (int jj = 0; jj < 8; jj++)
      {
        int i = (is + ii);
        int j = (js + jj);

        if (i >= ny || j >= ny || j < i)
        {
          continue;
        }

        sums[ii * 8 + jj] += normal[i * nx + x] * normal[j * nx + x];
      }
    }
  }

  for (int ii = 0; ii < 8; ii++)
  {
    for (int jj = 0; jj < 8; jj++)
    {
      int i = (is + ii);
      int j = (js + jj);

      if (i >= ny || j >= ny || j < i)
      {
        continue;
      }

      result[i * ny + j] = sums[ii * 8 + jj] * (nss[i] * nss[j]);
    }
  }
}

void correlate(int ny, int nx, const float *data, float *result)
{
  int y_parts = divup(ny, 8);
  int nyp = y_parts * 8;

  // Allocate device memory
  float *d_data, *d_normal, *d_nss, *d_result;

  size_t data_size = ny * nx * sizeof(float);
  size_t normal_size = nyp * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);
  size_t nss_size = ny * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_normal, normal_size));
  CHECK(hipMalloc((void **)&d_nss, nss_size));
  CHECK(hipMalloc((void **)&d_result, result_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_normal, 0, normal_size));
  CHECK(hipMemset(d_nss, 0, nss_size));
  CHECK(hipMemset(d_result, 0, result_size));

  // Copy input data to device
  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  // Launch kernel to compute means and normalize data
  {
    dim3 grid_dim(divup(ny, 8));
    dim3 block_dim(8);
    compute_means_and_normalize<<<grid_dim, block_dim>>>(ny, nx, d_data, d_normal, d_nss);
    CHECK(hipGetLastError());
  }

  // Launch kernel to compute correlations
  {
    dim3 grid_dim(divup(y_parts, 8), divup(y_parts, 8));
    dim3 block_dim(8, 8);
    compute_correlations<<<grid_dim, block_dim>>>(ny, nx, d_normal, d_nss, d_result);
    CHECK(hipGetLastError());
  }

  // Copy result back to host
  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  // Free device memory
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_normal));
  CHECK(hipFree(d_nss));
  CHECK(hipFree(d_result));
}
