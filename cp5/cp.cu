#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(x) check(x, #x)
static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

inline int static roundup(int a, int b)
{
  return divup(a, b) * b;
}

__global__ void compute_means_and_normalize(int ny, int nx, const float *data, float *normal, float *nss)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y >= ny)
  {
    return;
  }

  float sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    sum += data[y * nx + x];
  }
  float mean = sum / nx;

  float pow_sum = 0.0;
  for (int x = 0; x < nx; x++)
  {
    float normalized = data[y * nx + x] - mean;
    normal[x * ny + y] = normalized;
    pow_sum += normalized * normalized;
  }
  nss[y] = 1.0 / sqrt(pow_sum);
}

__global__ void compute_correlations(int ny, int nx, const float *normal, const float *nss, float *result)
{
  int ia = threadIdx.x;
  int ja = threadIdx.y;
  int ic = blockIdx.x;
  int jc = blockIdx.y;

  float sums[8][8] = {0};
  for (int k = 0; k < nx; k++)
  {
    float x[8];
    float y[8];

    for (int ijb = 0; ijb < 8; ijb++)
    {
      int i = ic * 64 + ijb * 8 + ia;
      int j = jc * 64 + ijb * 8 + ja;
      x[ijb] = normal[ny * k + i];
      y[ijb] = normal[ny * k + j];
    }

    for (int ib = 0; ib < 8; ib++)
    {
      for (int jb = 0; jb < 8; jb++)
      {
        sums[ib][jb] += x[ib] * y[jb];
      }
    }
  }

  for (int ib = 0; ib < 8; ib++)
  {
    for (int jb = 0; jb < 8; jb++)
    {
      int i = ic * 64 + ib * 8 + ia;
      int j = jc * 64 + jb * 8 + ja;
      if (i < ny && j < ny)
      {
        result[i + j * ny] = sums[ib][jb] * (nss[i] * nss[j]);
      }
    }
  }
}

void correlate(int ny, int nx, const float *data, float *result)
{
  int nn = roundup(ny, 64);

  // Allocate device memory
  float *d_data, *d_normal, *d_nss, *d_result;

  size_t data_size = ny * nx * sizeof(float);
  size_t normal_size = nn * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);
  size_t nss_size = ny * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_normal, normal_size));
  CHECK(hipMalloc((void **)&d_nss, nss_size));
  CHECK(hipMalloc((void **)&d_result, result_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_normal, 0, normal_size));
  CHECK(hipMemset(d_nss, 0, nss_size));
  CHECK(hipMemset(d_result, 0, result_size));

  // Copy input data to device
  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  // Launch kernel to compute means and normalize data
  {
    dim3 grid_dim(divup(ny, 8));
    dim3 block_dim(8);
    compute_means_and_normalize<<<grid_dim, block_dim>>>(ny, nx, d_data, d_normal, d_nss);
    CHECK(hipGetLastError());
  }

  // Launch kernel to compute correlations
  {
    dim3 grid_dim(nn / 64, nn / 64);
    dim3 block_dim(8, 8);
    compute_correlations<<<grid_dim, block_dim>>>(ny, nx, d_normal, d_nss, d_result);
    CHECK(hipGetLastError());
  }

  // Copy result back to host
  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  // Free device memory
  CHECK(hipFree(d_data));
  CHECK(hipFree(d_normal));
  CHECK(hipFree(d_nss));
  CHECK(hipFree(d_result));
}
